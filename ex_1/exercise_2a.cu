
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

int MAX_PARTICLES;
int NUM_ITERATIONS;
int TPB;
float DEC_FACTOR;
float TOLERANCE = 1e-6;

typedef struct {
    float3 position;
    float3 velocity;
} Particle;

__global__ void timestepGPU(Particle* array, int nPart, float dec_fact)
{
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    if (myId < nPart)
    {
        array[myId].velocity.x = array[myId].velocity.x * dec_fact;
        array[myId].velocity.y = array[myId].velocity.y * dec_fact;
        array[myId].velocity.z = array[myId].velocity.z * dec_fact;
        array[myId].position.x = array[myId].position.x + array[myId].velocity.x; 
        array[myId].position.y = array[myId].position.y + array[myId].velocity.y; 
        array[myId].position.z = array[myId].position.z + array[myId].velocity.z; 
    }
}

void timestepCPU(Particle* array)
{
    for (int i = 0; i < MAX_PARTICLES; i++)
    {
        array[i].velocity.x = array[i].velocity.x * DEC_FACTOR;
        array[i].velocity.y = array[i].velocity.y * DEC_FACTOR;
        array[i].velocity.z = array[i].velocity.z * DEC_FACTOR;
        array[i].position.x = array[i].position.x + array[i].velocity.x; 
        array[i].position.y = array[i].position.y + array[i].velocity.y; 
        array[i].position.z = array[i].position.z + array[i].velocity.z;
    }
}

int compare(Particle* x, Particle* y)
{
    int value = 1;
    for(int i = 0; i < MAX_PARTICLES && value; i++)
    {
        value = value & (x[i].position.x - y[i].position.x < TOLERANCE); 
        value = value & (x[i].position.y - y[i].position.y < TOLERANCE); 
        value = value & (x[i].position.z - y[i].position.z < TOLERANCE); 
        value = value & (x[i].velocity.x - y[i].velocity.x < TOLERANCE); 
        value = value & (x[i].velocity.y - y[i].velocity.y < TOLERANCE); 
        value = value & (x[i].velocity.z - y[i].velocity.z < TOLERANCE); 
    }
    return value;
}

void initArray(Particle* p)
{
    for(int i = 0; i < MAX_PARTICLES; i++)
    {
        p[i].position.x = (float) rand() / RAND_MAX;
        p[i].position.y = (float) rand() / RAND_MAX;
        p[i].position.z = (float) rand() / RAND_MAX;
        p[i].velocity.x = (float) rand() / RAND_MAX;
        p[i].velocity.y = (float) rand() / RAND_MAX;
        p[i].velocity.z = (float) rand() / RAND_MAX;

    }
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
 }

void setParameters(int argc, char** argv) 
{
    switch (argc)
    {
        case 5: DEC_FACTOR = atof(argv[4]);
        case 4: TPB = atoi(argv[3]);
        case 3: NUM_ITERATIONS = atoi(argv[2]);   
        case 2: MAX_PARTICLES = atoi(argv[1]); break;
        default: MAX_PARTICLES = 100000; NUM_ITERATIONS = 100; TPB = 256; DEC_FACTOR = 0.9; break; 
    }
}

int main(int argc, char **argv)
{
    setParameters(argc, argv);
    
    //Input parametres:
    //[1] : Number of particles
    //[2] : Number of iterations
    //[3] : Decreasing factor of velocity (optional)

    double iStart, iElapsCPU, iElapsGPU;
    
    //Initialization of pointers
    Particle* pOriginal = (Particle*) malloc(MAX_PARTICLES * sizeof(Particle));
    initArray(pOriginal);

    Particle* pCPU = (Particle*) malloc(MAX_PARTICLES * sizeof(Particle));
    memcpy(pCPU, pOriginal, MAX_PARTICLES * sizeof(Particle));

    //Particle* pForeign = (Particle*) malloc(MAX_PARTICLES * sizeof(Particle));
    Particle* pForeign; 
    hipHostAlloc(&pForeign, MAX_PARTICLES * sizeof(Particle), hipHostMallocDefault);
    memcpy(pForeign, pOriginal, MAX_PARTICLES * sizeof(Particle));

    Particle* pGPU;
    hipMalloc(&pGPU, MAX_PARTICLES * sizeof(Particle));

    //Computing by CPU
    //printf("Computing by CPU... ");
    iStart = cpuSecond();
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        timestepCPU(pCPU);
    }
    iElapsCPU = cpuSecond() - iStart;
    //printf("Done\n");
    
    //Computing by GPU
    //printf("Computing by GPU... ");
    iStart = cpuSecond();
    for (int i = 0; i < NUM_ITERATIONS; i++)
    {
        //Moving data to the device
        hipMemcpy(pGPU, pForeign, MAX_PARTICLES * sizeof(Particle), hipMemcpyHostToDevice);   
        timestepGPU<<<(MAX_PARTICLES + TPB - 1)/TPB, TPB>>>(pGPU, MAX_PARTICLES, DEC_FACTOR);
        hipMemcpy(pForeign, pGPU, MAX_PARTICLES*sizeof(Particle), hipMemcpyDeviceToHost);
    }
    hipDeviceSynchronize();
    iElapsGPU = cpuSecond() - iStart;
    //printf("Done\n");

    //Sum up
    printf("\nSize of the array: %d\nTPB: %d\n", MAX_PARTICLES, TPB);
    printf("CPU time: %2f\nGPU time: %2f\n", iElapsCPU, iElapsGPU);


    int comp = compare(pForeign, pCPU);

    if (comp)
    {
        //printf("Both arrays are equal\n");
    }
    else 
    {
        printf("Differences between arrays\n");
    }
    return 0;
}